#include "hip/hip_runtime.h"
#include "cu_utils.h"
#include <iostream>

__global__ void cudaUtilities::__kernel__ewxty(int n, float* x, float* y, float* z)
{
    int tid;
    int off;
    int idx;
    tid = threadIdx.x + blockIdx.x * blockDim.x;
    off = blockDim.x * gridDim.x;

    idx = tid;
    while(idx < n) 
    {
        z[idx] = x[idx] * y[idx];
        idx += off;
    }
}

void cudaUtilities::ewxty(int n, float* x, float* y, hipStream_t stream, float *z)
{
    dim3 gridcfg(BLOCKS_PER_GRID_X, BLOCKS_PER_GRID_Y, 1);
    dim3 blockcfg(THREADS_PER_BLOCK_X, THREADS_PER_BLOCK_Y, 1);

    __kernel__ewxty<<<gridcfg, blockcfg, 0, stream>>>(n, x, y, z);

    /*
    hipDeviceSynchronize();
    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
    {
        std::cerr << "Error in kernel launch: ";
        std::cerr << hipGetErrorString(err) << std::endl;
        exit(-1);
    }
    */ 
}