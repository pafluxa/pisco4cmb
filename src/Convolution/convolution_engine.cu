#include "hip/hip_runtime.h"
#include "convolution_engine.h"

#include "cu_utils.h"
#include "Sphtrigo/sphtrigo.hpp"

// healpix stuff
#include <arr.h>
#include <pointing.h>
#include <healpix_base.h>
// for OpenMP
#include <omp.h>

#include <iostream>
#include <iomanip>

// Constructor
ConvolutionEngine::ConvolutionEngine(int nsideSky, int nsideBeam, int nSamples) {

    int i;    
    this->nSamples = nSamples;
    this->nsideSky = nsideSky;
    this->nsideBeam = nsideBeam;
    nspix = 12 * nsideSky * nsideSky;
    nbpix = 12 * nsideBeam * nsideBeam;
    nnz = 4 * nspix;
    
    // create cuSparse context
    CATCH_CUSPARSE( hipsparseCreate(&cuspH) )

    // create cuBLAS context
    CATCH_CUBLAS(hipblasCreate(&cublasH))

    // create streams for transfer
    for(i = 0; i < N_TRANSFER_STREAMS; i++)
    {
        CATCH_CUDA(hipStreamCreate(&transferStreams[i]))
    }
    // create streams for processing
    for(i = 0; i < N_PROC_STREAMS; i++)
    {
        CATCH_CUDA(hipStreamCreate(&procStreams[i]))
    }
}

// dummy destructor
ConvolutionEngine::~ConvolutionEngine(void)
{
    int i;

    // clean-up CUBLAS context
    CATCH_CUBLAS(hipblasDestroy(cublasH))
    
    // clean-up cuSparse context
    CATCH_CUSPARSE( hipsparseDestroy(cuspH))

    // destroy transfer streams
    // create streams for transfer
    for(i = 0; i < N_TRANSFER_STREAMS; i++)
    {
        CATCH_CUDA(hipStreamDestroy(transferStreams[i]))
    }
    
    for(i = 0; i < N_PROC_STREAMS; i++)
    {
        CATCH_CUDA(hipStreamDestroy(procStreams[i]))
    }
    
}

void ConvolutionEngine::allocate_host_buffers(void) {
    // matrix buffers are allocated using pinned memory 
    // because they are constantly updated
    size_t bytes;
    // allocate buffers for matrix
    bytes = sizeof(float) * nnz;
    CATCH_CUDA(hipHostMalloc((void**)&csrValR, bytes))
    bytes = sizeof(int) * nnz;
    CATCH_CUDA(hipHostMalloc((void**)&csrColIndR, bytes))
    bytes = sizeof(int) * (nspix + 1);
    CATCH_CUDA(hipHostMalloc((void**)&csrRowPtrR, bytes))
    // allocate bufferse for cosine and sine 2 chi
    bytes = sizeof(float) * nspix;
    CATCH_CUDA(hipHostMalloc((void**)&s2chi, bytes))
    CATCH_CUDA(hipHostMalloc((void**)&c2chi, bytes))
    // sky pixel coordinates
    raPixSky = (float *)malloc(sizeof(float) * nspix);
    decPixSky = (float *)malloc(sizeof(float) * nspix);
}

void ConvolutionEngine::allocate_device_buffers(void) {
    
    float alpha; 
    float beta;
    float addme;
    float subme;
    
    size_t bytes;
    
    // allocate buffers for sky
    bytes = sizeof(float) * nspix;
    CATCH_CUDA(hipMalloc((void**)&dev_stokesI, bytes))
    CATCH_CUDA(hipMalloc((void**)&dev_stokesQ, bytes))
    CATCH_CUDA(hipMalloc((void**)&dev_stokesU, bytes))
    CATCH_CUDA(hipMalloc((void**)&dev_stokesV, bytes))
    // allocate buffers for beams
    bytes = sizeof(float) * nbpix;
    CATCH_CUDA(hipMalloc((void**)&dev_beamIa, bytes))
    CATCH_CUDA(hipMalloc((void**)&dev_beamQa, bytes))
    CATCH_CUDA(hipMalloc((void**)&dev_beamUa, bytes))
    CATCH_CUDA(hipMalloc((void**)&dev_beamVa, bytes))
    CATCH_CUDA(hipMalloc((void**)&dev_beamIb, bytes))
    CATCH_CUDA(hipMalloc((void**)&dev_beamQb, bytes))
    CATCH_CUDA(hipMalloc((void**)&dev_beamUb, bytes))
    CATCH_CUDA(hipMalloc((void**)&dev_beamVb, bytes))
    // allocate buffers for aligned beams
    bytes = sizeof(float) * nspix;
    CATCH_CUDA(hipMalloc((void**)&dev_AbeamIa, bytes))
    CATCH_CUDA(hipMalloc((void**)&dev_AbeamQa, bytes))
    CATCH_CUDA(hipMalloc((void**)&dev_AbeamUa, bytes))
    CATCH_CUDA(hipMalloc((void**)&dev_AbeamVa, bytes))
    CATCH_CUDA(hipMalloc((void**)&dev_AbeamIb, bytes))
    CATCH_CUDA(hipMalloc((void**)&dev_AbeamQb, bytes))
    CATCH_CUDA(hipMalloc((void**)&dev_AbeamUb, bytes))
    CATCH_CUDA(hipMalloc((void**)&dev_AbeamVb, bytes))
    // allocate buffers for cosine and sine 2 chi
    bytes = sizeof(float) * nspix;
    CATCH_CUDA(hipMalloc((void**)&dev_s2chi, bytes))
    CATCH_CUDA(hipMalloc((void**)&dev_c2chi, bytes))
    // allocate buffers for matrix
    bytes = sizeof(float) * nnz;
    CATCH_CUDA(hipMalloc((void**)&dev_csrValR, bytes))
    bytes = sizeof(int) * nnz;
    CATCH_CUDA(hipMalloc((void**)&dev_csrColIndR, bytes))
    bytes = sizeof(int) * (nspix + 1);
    CATCH_CUDA(hipMalloc((void**)&dev_csrRowPtrR, bytes))
    // allocate buffers for calculation of D
    bytes = sizeof(float) * nspix;
    CATCH_CUDA(hipMalloc((void**)&dev_DQ1a, bytes))
    CATCH_CUDA(hipMalloc((void**)&dev_DQ2a, bytes))
    CATCH_CUDA(hipMalloc((void**)&dev_DU1a, bytes))
    CATCH_CUDA(hipMalloc((void**)&dev_DU2a, bytes))
    CATCH_CUDA(hipMalloc((void**)&dev_DQ1b, bytes))
    CATCH_CUDA(hipMalloc((void**)&dev_DQ2b, bytes))
    CATCH_CUDA(hipMalloc((void**)&dev_DU1b, bytes))
    CATCH_CUDA(hipMalloc((void**)&dev_DU2b, bytes))
    /* detector data */
    bytes = sizeof(float) * nSamples;
    CATCH_CUDA(hipMalloc((void**)&dev_ia, bytes))
    CATCH_CUDA(hipMalloc((void**)&dev_ib, bytes))
    CATCH_CUDA(hipMalloc((void**)&dev_qa, bytes))
    CATCH_CUDA(hipMalloc((void**)&dev_qb, bytes))
    CATCH_CUDA(hipMalloc((void**)&dev_ua, bytes))
    CATCH_CUDA(hipMalloc((void**)&dev_ub, bytes))
    /* cuda uses device memory for EVERYTHING*/

    alpha = 1.0;
    beta = 0.0;
    addme = 1.0;
    subme = -1.0;

    // cuBLAS operates in device mode, so we need to make explicit copies
    // of constants from the host to the device.    
    CATCH_CUDA( hipMalloc((void **)&dev_alpha, sizeof(float)) )
    CATCH_CUDA( hipMalloc((void **)&dev_beta, sizeof(float)) )
    CATCH_CUDA( hipMalloc((void **)&dev_addme, sizeof(float)) )
    CATCH_CUDA( hipMalloc((void **)&dev_subme, sizeof(float)) )
    CATCH_CUDA( hipMemcpy(dev_alpha, &alpha, sizeof(float), hipMemcpyHostToDevice))
    CATCH_CUDA( hipMemcpy(dev_beta, &beta, sizeof(float), hipMemcpyHostToDevice))
    CATCH_CUDA( hipMemcpy(dev_addme, &addme, sizeof(float), hipMemcpyHostToDevice))
    CATCH_CUDA( hipMemcpy(dev_subme, &subme, sizeof(float), hipMemcpyHostToDevice))
}

// creates internal cuSparse CSR matrix. 
void ConvolutionEngine::create_matrix(void) {
    
    CATCH_CUSPARSE(
        hipsparseCreateCsr(
            &R, nspix, nbpix, nnz,
            dev_csrRowPtrR, dev_csrColIndR, dev_csrValR,
            HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
            HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F)
    )
}

/* creates internal cuSparse object reprensenting the beam. */
void ConvolutionEngine::beam_to_cuspvec(PolBeam* beam) {
    size_t beamBufferSize; 
    /* size (in bytes) of buffer to be copied. */
    beamBufferSize = sizeof(float) * nbpix;
    /* copy data to device for detector a. */
    CATCH_CUDA(
        hipMemcpy(dev_beamIa, beam->get_I_beam('a'), beamBufferSize, hipMemcpyHostToDevice))
    CATCH_CUDA(
        hipMemcpy(dev_beamQa, beam->get_Q_beam('a'), beamBufferSize, hipMemcpyHostToDevice))
    CATCH_CUDA(
        hipMemcpy(dev_beamUa, beam->get_U_beam('a'), beamBufferSize, hipMemcpyHostToDevice))
    CATCH_CUDA(
        hipMemcpy(dev_beamVa, beam->get_V_beam('a'), beamBufferSize, hipMemcpyHostToDevice))
    /* copy data to device for detector b. */
    CATCH_CUDA(
        hipMemcpy(dev_beamIb, beam->get_I_beam('b'), beamBufferSize, hipMemcpyHostToDevice))
    CATCH_CUDA(
        hipMemcpy(dev_beamQb, beam->get_Q_beam('b'), beamBufferSize, hipMemcpyHostToDevice))
    CATCH_CUDA(
        hipMemcpy(dev_beamUb, beam->get_U_beam('b'), beamBufferSize, hipMemcpyHostToDevice))
    CATCH_CUDA(
        hipMemcpy(dev_beamVb, beam->get_V_beam('b'), beamBufferSize, hipMemcpyHostToDevice))
    
    CATCH_CUSPARSE(hipsparseCreateDnVec(&beamIa, nbpix, dev_beamIa, HIP_R_32F))
    CATCH_CUSPARSE(hipsparseCreateDnVec(&beamQa, nbpix, dev_beamQa, HIP_R_32F))
    CATCH_CUSPARSE(hipsparseCreateDnVec(&beamUa, nbpix, dev_beamUa, HIP_R_32F))
    CATCH_CUSPARSE(hipsparseCreateDnVec(&beamVa, nbpix, dev_beamVa, HIP_R_32F))
    CATCH_CUSPARSE(hipsparseCreateDnVec(&beamIb, nbpix, dev_beamIb, HIP_R_32F))
    CATCH_CUSPARSE(hipsparseCreateDnVec(&beamQb, nbpix, dev_beamQb, HIP_R_32F))
    CATCH_CUSPARSE(hipsparseCreateDnVec(&beamUb, nbpix, dev_beamUb, HIP_R_32F))
    CATCH_CUSPARSE(hipsparseCreateDnVec(&beamVb, nbpix, dev_beamVb, HIP_R_32F))
    
    CATCH_CUSPARSE(hipsparseCreateDnVec(&AbeamIa, nspix, dev_AbeamIa, HIP_R_32F))
    CATCH_CUSPARSE(hipsparseCreateDnVec(&AbeamQa, nspix, dev_AbeamQa, HIP_R_32F))
    CATCH_CUSPARSE(hipsparseCreateDnVec(&AbeamUa, nspix, dev_AbeamUa, HIP_R_32F))
    CATCH_CUSPARSE(hipsparseCreateDnVec(&AbeamVa, nspix, dev_AbeamVa, HIP_R_32F))
    CATCH_CUSPARSE(hipsparseCreateDnVec(&AbeamIb, nspix, dev_AbeamIb, HIP_R_32F))
    CATCH_CUSPARSE(hipsparseCreateDnVec(&AbeamQb, nspix, dev_AbeamQb, HIP_R_32F))
    CATCH_CUSPARSE(hipsparseCreateDnVec(&AbeamUb, nspix, dev_AbeamUb, HIP_R_32F))
    CATCH_CUSPARSE(hipsparseCreateDnVec(&AbeamVb, nspix, dev_AbeamVb, HIP_R_32F))


}

/* creates internal cuSparse object reprensenting the sky. */
void ConvolutionEngine::sky_to_cuspvec(Sky* sky) {
    size_t bufferSize; 
    /* size (in bytes) of buffer to be copied. */
    bufferSize = sizeof(float) * nspix;
    /* copy data to device for detector a. */
    CATCH_CUDA(
        hipMemcpy(dev_stokesI, sky->get_I(), bufferSize, hipMemcpyHostToDevice))
    CATCH_CUDA(
        hipMemcpy(dev_stokesQ, sky->get_Q(), bufferSize, hipMemcpyHostToDevice))
    CATCH_CUDA(
        hipMemcpy(dev_stokesU, sky->get_U(), bufferSize, hipMemcpyHostToDevice))
    CATCH_CUDA(
        hipMemcpy(dev_stokesV, sky->get_V(), bufferSize, hipMemcpyHostToDevice))
    // create dense vector descriptors for cuSparse 
    CATCH_CUSPARSE(hipsparseCreateDnVec(&stokesI, nspix, dev_stokesI, HIP_R_32F))
    CATCH_CUSPARSE(hipsparseCreateDnVec(&stokesQ, nspix, dev_stokesQ, HIP_R_32F))
    CATCH_CUSPARSE(hipsparseCreateDnVec(&stokesU, nspix, dev_stokesU, HIP_R_32F))
    CATCH_CUSPARSE(hipsparseCreateDnVec(&stokesV, nspix, dev_stokesV, HIP_R_32F))
}



void ConvolutionEngine::data_to_host(float *a, float *b)
{
    int i;
    float *ia;
    float *qa;
    float *ua;
    float *ib;
    float *qb;
    float *ub;

    ia = (float *)malloc(sizeof(float) * nSamples);
    qa = (float *)malloc(sizeof(float) * nSamples);
    ua = (float *)malloc(sizeof(float) * nSamples);
    ib = (float *)malloc(sizeof(float) * nSamples);
    qb = (float *)malloc(sizeof(float) * nSamples);
    ub = (float *)malloc(sizeof(float) * nSamples);

    CATCH_CUDA(hipMemcpy(ia, dev_ia, sizeof(float) * nSamples, hipMemcpyDeviceToHost))
    CATCH_CUDA(hipMemcpy(qa, dev_qa, sizeof(float) * nSamples, hipMemcpyDeviceToHost))
    CATCH_CUDA(hipMemcpy(ua, dev_ua, sizeof(float) * nSamples, hipMemcpyDeviceToHost))
    CATCH_CUDA(hipMemcpy(ib, dev_ib, sizeof(float) * nSamples, hipMemcpyDeviceToHost))
    CATCH_CUDA(hipMemcpy(qb, dev_qb, sizeof(float) * nSamples, hipMemcpyDeviceToHost))
    CATCH_CUDA(hipMemcpy(ub, dev_ub, sizeof(float) * nSamples, hipMemcpyDeviceToHost))

    for(i = 0; i < nSamples; i++)
    {
        a[i] = ia[i] + qa[i] + ua[i];
        b[i] = ib[i] + qb[i] + ub[i];
    }

    free(ia);
    free(qa);
    free(ua);
    free(ib);
    free(qb);
    free(ub);
}

void ConvolutionEngine::calculate_sky_pixel_coordinates(Sky* sky) {

    int npixels;
    int skyPix;
    pointing sp;
    
    npixels = 12 * nsideSky * nsideSky;
    for(skyPix = 0; skyPix < npixels; skyPix++)
    {
        sp = sky->hpxBase.pix2ang(skyPix);
        
        raPixSky[skyPix] = float(sp.phi);
        decPixSky[skyPix] = float(M_PI_2 - sp.theta);
    }
}

/* creates internal cuSparse object reprensenting the "rotated" beam. */
void ConvolutionEngine::update_matrix(int pix, float* weights, int* neigh) {

    csrValR[4 * pix + 0] = weights[0];    
    csrValR[4 * pix + 1] = weights[1];    
    csrValR[4 * pix + 2] = weights[2];    
    csrValR[4 * pix + 3] = weights[3];    

    csrColIndR[4 * pix + 0] = neigh[0];
    csrColIndR[4 * pix + 1] = neigh[1];
    csrColIndR[4 * pix + 2] = neigh[2];
    csrColIndR[4 * pix + 3] = neigh[3];

    csrRowPtrR[pix] = 4 * pix;
}

void ConvolutionEngine::update_chi(int pix, float chi) {

    s2chi[pix] = sinf(2.0f * chi);
    c2chi[pix] = cosf(2.0f * chi);
}

void ConvolutionEngine::fill_matrix(Sky* sky, PolBeam* beam, 
    float ra_bc, float dec_bc, float psi_bc) {

    #pragma omp parallel
    {
    int i;
    int idx;
    int off;
    int skyPix;
    int chunksize;
    int reminder;
    double ra_pix;
    double dec_pix;
    double rho;
    double sigma;
    double chi;
    pointing bp;
    fix_arr<int, 4> neigh;
    fix_arr<double, 4> wgh;
    float* weights;
    int* neighbors;
    int* pixels;
    
    idx = omp_get_thread_num();
    off = omp_get_num_threads();
    chunksize = nspix / off;
    // last thread gets more work
    reminder = 0;
    if(idx == omp_get_thread_num() - 1) {
        reminder += (nspix - chunksize * off);
    }
    // allocate local (thread-safe) memory space to store results
    pixels = (int *)malloc(sizeof(int) * (chunksize + reminder));
    neighbors = (int *)malloc(sizeof(int) * 4 * (chunksize + reminder));
    weights = (float *)malloc(sizeof(float) * 4 * (chunksize + reminder));
    
    // Calculate equivalent beam coordinates for every sky pixel. 
    i = 0; 
    while(i < chunksize + reminder)
    {
        skyPix = idx * chunksize + i;        
        // coordinates of sky pixel already computed. 
        ra_pix = raPixSky[skyPix];
        dec_pix = decPixSky[skyPix];

        // get equivalent beam coordinates and polarization angle. 
        SphericalTransformations::rho_sigma_chi_pix(
            &rho, &sigma, &chi,
            double(ra_bc), double(dec_bc), double(psi_bc),
            ra_pix, dec_pix);

        // get interpolation information for beam at (rho, sigma). 
        bp.theta = double(rho);
        bp.phi = double(sigma);
        // handle bug in acos
        if(bp.theta >= M_PI) 
        { 
            bp.theta = double(M_PI); 
        }
        beam->hpxBase.get_interpol(bp, neigh, wgh);

        weights[4*i + 0] = float(wgh[0]);    
        weights[4*i + 1] = float(wgh[1]);    
        weights[4*i + 2] = float(wgh[2]);    
        weights[4*i + 3] = float(wgh[3]);
        neighbors[4*i + 0] = neigh[0];    
        neighbors[4*i + 1] = neigh[1];    
        neighbors[4*i + 2] = neigh[2];    
        neighbors[4*i + 3] = neigh[3];   

        pixels[i] = skyPix;

        i++;
    }

    // every thread performs memcpy in parallel
    memcpy(csrValR + idx * 4 * chunksize, weights, sizeof(float) * (chunksize + reminder) * 4);
    memcpy(csrColIndR + idx * 4 * chunksize, neighbors, sizeof(int) * (chunksize + reminder) * 4);
    memcpy(csrRowPtrR + idx * chunksize, pixels, sizeof(int) * (chunksize + reminder));
    
    free(weights);
    free(neighbors);
    free(pixels);
    /*
    {
        // update matrix values 
        update_matrix(skyPix, weights, neighbors);
        update_chi(skyPix, chi);

        i++;
    }
    */
    }
}

/* copies sine and consine chi and matrix to device.*/
void ConvolutionEngine::exec_transfer(void) {

    size_t bytes;

    // synchronize device to avoid writing while computing
    CATCH_CUDA( hipDeviceSynchronize() )

    // fill last value of csrRowPtrR
    csrRowPtrR[nspix] = 4 * nspix;

    // transfer cosine and sine of 2 chi
    bytes = sizeof(float) * nspix;
    CATCH_CUDA(hipMemcpy(dev_s2chi, s2chi, bytes, hipMemcpyHostToDevice))
    CATCH_CUDA(hipMemcpy(dev_c2chi, c2chi, bytes, hipMemcpyHostToDevice))
    
    // transfer matrix descriptors (CSR format)
    bytes = sizeof(float) * nnz;
    CATCH_CUDA(hipMemcpy(dev_csrValR, csrValR, bytes, hipMemcpyHostToDevice))
    bytes = sizeof(int) * nnz;
    CATCH_CUDA(hipMemcpy(dev_csrColIndR, csrColIndR, bytes, hipMemcpyHostToDevice))
    bytes = sizeof(int) * (nspix + 1);
    CATCH_CUDA(hipMemcpy(dev_csrRowPtrR, csrRowPtrR, bytes, hipMemcpyHostToDevice))
}

void ConvolutionEngine::exec_single_convolution_step(int s) {

    size_t bufferSize;

    // set cuSparse to run in device mode so all memory operations
    // happen in the GPU (fully async)
    CATCH_CUSPARSE(hipsparseSetPointerMode(cuspH, HIPSPARSE_POINTER_MODE_DEVICE))

    // set CUBLAS to run in device mode, i.e., all memory operations happen 
    // in the GPU
    CATCH_CUBLAS( hipblasSetPointerMode(cublasH, HIPBLAS_POINTER_MODE_DEVICE) )

    // calculate required extra space and allocate if needed 
    CATCH_CUSPARSE(
        hipsparseSpMV_bufferSize(
            cuspH, HIPSPARSE_OPERATION_NON_TRANSPOSE,
            dev_alpha, R, beamIa, dev_beta, AbeamIa, HIP_R_32F,
            HIPSPARSE_SPMV_CSR_ALG2, &bufferSize)
    )
    
    if(bufferSize != cusparse_buffersize)
    {
        if(cusparse_buffersize != 0)
        {
            CATCH_CUDA( hipFree(dev_tempBuffer) )
        }
        cusparse_buffersize = bufferSize;
        CATCH_CUDA( hipMalloc(&dev_tempBuffer, cusparse_buffersize) )
    }

    // capture graph execution
    if(useGraph)
    {
        CATCH_CUDA( hipGraphLaunch(graph_exec, procStreams[0]) )
    }
    else
    {
        CATCH_CUSPARSE( hipsparseSetStream(cuspH, procStreams[0]) )
        CATCH_CUDA( hipStreamBeginCapture(procStreams[0], hipStreamCaptureModeGlobal) )
        // Detector a
        // calculate R x beam for Stokes I beam of detector a
        CATCH_CUSPARSE(
            hipsparseSpMV(cuspH,
                HIPSPARSE_OPERATION_NON_TRANSPOSE, 
                dev_alpha, R, beamIa, dev_beta, AbeamIa,
                HIP_R_32F,
                HIPSPARSE_SPMV_CSR_ALG2, 
                dev_tempBuffer)
        )

        // Stokes Q beam of detector a ...
        CATCH_CUSPARSE(
            hipsparseSpMV(cuspH,
                HIPSPARSE_OPERATION_NON_TRANSPOSE,
                dev_alpha, R, beamQa, dev_beta, AbeamQa,
                HIP_R_32F,
                HIPSPARSE_SPMV_CSR_ALG2, 
                dev_tempBuffer)
        )
        // Stokes U beam of detector a
        CATCH_CUSPARSE(
            hipsparseSpMV(cuspH,
                HIPSPARSE_OPERATION_NON_TRANSPOSE, 
                dev_alpha, R, beamUa, dev_beta, AbeamUa,
                HIP_R_32F,
                HIPSPARSE_SPMV_CSR_ALG2, 
                dev_tempBuffer)
        )
        // Repeat for detector b 
        // calculate R x beam for:
        // - Stokes I beam of detector b
        CATCH_CUSPARSE(
            hipsparseSpMV(cuspH,
                HIPSPARSE_OPERATION_NON_TRANSPOSE, 
                dev_alpha, R, beamIb, dev_beta, AbeamIb,
                HIP_R_32F,
                HIPSPARSE_SPMV_CSR_ALG2, 
                dev_tempBuffer)
        )
        // - Stokes Q beam of detector b ...
        CATCH_CUSPARSE(
            hipsparseSpMV(cuspH,
                HIPSPARSE_OPERATION_NON_TRANSPOSE, 
                dev_alpha, R, beamQb, dev_beta, AbeamQb,
                HIP_R_32F,
                HIPSPARSE_SPMV_CSR_ALG2, 
                dev_tempBuffer)
        )
        // - Stokes U beam of detector b
        CATCH_CUSPARSE(
            hipsparseSpMV(cuspH,
                HIPSPARSE_OPERATION_NON_TRANSPOSE, 
                dev_alpha, R, beamUb, dev_beta, AbeamUb,
                HIP_R_32F,
                HIPSPARSE_SPMV_CSR_ALG2, 
                dev_tempBuffer)
        )
        CATCH_CUDA(hipStreamEndCapture(procStreams[0], &graph))
        CATCH_CUDA(hipDeviceSynchronize())
        CATCH_CUDA(hipGetLastError())
        CATCH_CUDA(hipGraphInstantiateWithFlags(&graph_exec, graph, 0))
        useGraph = true;
    }
    //CATCH_CUDA( hipStreamSynchronize(procStreams[0]) )

    // calculate D_Q for detector a
    // note: the order in which operations below are executed should not be changed
    //       as it was designed to maximize concurrency with cuSparse
    // - multiply cos(2chi) by Q beam
    cudaUtilities::ewxty(nspix, dev_AbeamQa, dev_c2chi, procStreams[0], dev_DQ1a); 
    // - multiply sin(2chi) by Q beam
    cudaUtilities::ewxty(nspix, dev_AbeamQa, dev_s2chi, procStreams[0], dev_DU2a);
    // - multiply sin(2chi) by U beam
    cudaUtilities::ewxty(nspix, dev_AbeamUa, dev_s2chi, procStreams[0], dev_DQ2a);
    // - multiply cos(2chi) by U beam
    cudaUtilities::ewxty(nspix, dev_AbeamUa, dev_c2chi, procStreams[0], dev_DU1a);
    // - multiply cos(2chi) by Q beam
    cudaUtilities::ewxty(nspix, dev_AbeamQb, dev_c2chi, procStreams[0], dev_DQ1b);
    // - multiply sin(2chi) by Q beam
    cudaUtilities::ewxty(nspix, dev_AbeamQb, dev_s2chi, procStreams[0], dev_DU2b);
    // - multiply sin(2chi) by U beam
    cudaUtilities::ewxty(nspix, dev_AbeamUb, dev_s2chi, procStreams[0], dev_DQ2b);
    // - multiply cos(2chi) by U beam
    cudaUtilities::ewxty(nspix, dev_AbeamUb, dev_c2chi, procStreams[0], dev_DU1b);

    // calculate D for detector a 
    // - add and overwrite DQ1
    CATCH_CUBLAS( hipblasSetStream(cublasH, procStreams[0]) )
    CATCH_CUBLAS(hipblasSaxpy(cublasH, nspix, dev_subme, dev_DQ2a, 1, dev_DQ1a, 1))
    // - add and overwrite DU1
    CATCH_CUBLAS(hipblasSaxpy(cublasH, nspix, dev_addme, dev_DU2a, 1, dev_DU1a, 1))

    // calculate D for detector b 
    // - scale DQ1b by -1.0
    CATCH_CUBLAS(hipblasSscal(cublasH, nspix, dev_subme, dev_DQ1b, 1))
    // - add and overwrite DQ1b = -(Q c2chi) + (U s2chi)
    CATCH_CUBLAS(hipblasSaxpy(cublasH, nspix, dev_addme, dev_DQ2b, 1, dev_DQ1b, 1))
    // - scale DU1b by -1.0 
    CATCH_CUBLAS(hipblasSscal(cublasH, nspix, dev_subme, dev_DU1b, 1))
    // - scale DU2 by -1.0 
    CATCH_CUBLAS(hipblasSscal(cublasH, nspix, dev_subme, dev_DU2b, 1))
    // - add and overwrite DU1 = - (U c2chi + Q s2chi)
    CATCH_CUBLAS(hipblasSaxpy(cublasH, nspix, dev_addme, dev_DU2b, 1, dev_DU1b, 1))
    
    // convolution is now a dot product
    // - Stokes I part for detector a 
    CATCH_CUBLAS(hipblasSdot(cublasH, nspix, dev_stokesI, 1, dev_AbeamIa, 1, dev_ia + s));
    // - Stokes Q part for detector a
    CATCH_CUBLAS(hipblasSdot(cublasH, nspix, dev_stokesQ, 1, dev_DQ1a, 1, dev_qa + s));
    // - Stokes U part for detector a
    CATCH_CUBLAS(hipblasSdot(cublasH, nspix, dev_stokesU, 1, dev_DU1a, 1, dev_ua + s));
    
    // - Stokes I part for detector b 
    CATCH_CUBLAS(hipblasSdot(cublasH, nspix, dev_stokesI, 1, dev_AbeamIb, 1, dev_ib + s));
    // - Stokes Q part for detector a
    CATCH_CUBLAS(hipblasSdot(cublasH, nspix, dev_stokesQ, 1, dev_DQ1b, 1, dev_qb + s));
    // - Stokes U part for detector a
    CATCH_CUBLAS(hipblasSdot(cublasH, nspix, dev_stokesU, 1, dev_DU1b, 1, dev_ub + s));
}